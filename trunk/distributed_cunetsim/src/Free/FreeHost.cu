/*
 * FreeHost.cu
 *
 *  Created on: Dec 7, 2011
 *      Author: benromdh
 */


#ifndef STRUCTURES_H_
#define STRUCTURES_H_
#include "../structures.h"
#endif /* STRUCTURES_H_ */


__host__ void Free_Host(struct Geo *Host_Geo, struct Geo2 *Host_Geo2, struct Cell *Host_Cell, struct Node *Host_Node, struct Buffer *Host_InPhy, struct Buffer *Host_OutPhy, struct RouterBuffer *Host_RouterBuffer, float *Host_PosRandx, float  *Host_PosRandy, float *Host_PosRandz, float *Host_VRandx, float  *Host_VRandy, float *Host_VRandz)
{
	hipHostFree(Host_Geo);
	hipHostFree(Host_Geo2);
	hipHostFree(Host_Cell);
	hipHostFree(Host_Node);
	hipHostFree(Host_InPhy);
	hipHostFree(Host_OutPhy);
	hipHostFree(Host_RouterBuffer);
	hipHostFree(Host_PosRandx);
	hipHostFree(Host_PosRandy);
	hipHostFree(Host_PosRandz);
	hipHostFree(Host_VRandx);
	hipHostFree(Host_VRandy);
	hipHostFree(Host_VRandz);
}
