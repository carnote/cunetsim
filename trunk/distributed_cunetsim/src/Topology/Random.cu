#include "hip/hip_runtime.h"
/*******************************************************************************

  Eurecom Cunetsim2
  Copyright(c) 2011 - 2012 Eurecom

  This program is free software; you can redistribute it and/or modify it
  under the terms and conditions of the GNU General Public License,
  version 2, as published by the Free Software Foundation.

  This program is distributed in the hope it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for
  more details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc.,
  51 Franklin St - Fifth Floor, Boston, MA 02110-1301 USA.

  The full GNU General Public License is included in this distribution in
  the file called "COPYING".

  Contact Information
  Cunetsim Admin: cunetsim@eurecom.fr
  Cunetsim Tech : cunetsim_tech@eurecom.fr
  Forums       : TODO
  Address      : Eurecom, 2229, route des crêtes, 06560 Valbonne Sophia Antipolis, France

*******************************************************************************/

#ifndef STRUCTURES_H_
#define STRUCTURES_H_
#include "../structures.h"
#include "/usr/local/cuda/include/hiprand.h"
#endif /* STRUCTURES_H_ */
#ifndef INTERFACES_H_
#define INTERFACES_H_
#include "../interfaces.h"
#endif /* INTERFACES_H_ */
#include "../vars.h"

#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <mpi.h>

#define __timing__

extern float kernel_duration[1000][17];


struct Performances Random_sched(struct Partition partition, int f, float drop_prob,
		void *genV, int gui, MPI_Comm comm_masters, MPI_data_types mpi_types) {
	
#ifdef __timing__
	hipEvent_t start1, stop1, start2, stop2;// timers to compute teh runtime

	// Database variables
	MYSQL *conn = NULL;
	char *host_name = "pyroclaste";
  	char *user_name = "root";
  	char *password  = "database";
 	unsigned int port_num = 0;
  	char *socket_name = NULL;
  	char *db_name = "cunetsim";
  	int flags = 0;
	char query[4096];

#endif


	struct Cell *host_cell, *device_cell;
	struct Connection *device_ext_connections;
	float *host_pos_randx, *host_pos_randy, *host_pos_randz, *host_v_randx, *host_v_randy, *host_v_randz;
	float *device_randx, *device_randy, *device_randz, *device_randv;
	struct Performances performance;
	struct Geo *host_geo, *device_geo;
	struct Buffer *host_out_phy, *host_in_phy, *device_out_phy, *device_in_phy;
	struct RemoteCommBuffer *host_remote_comm_out, *host_remote_comm_in, *device_remote_comm_out, *device_remote_comm_in;
	struct MasterBuffer *master_in, *master_out;
	int *host_total_dest, *device_total_dest, *device_forwarded_per_node, *device_forwarded_per_node_out,
			*host_forwarded_per_node, *host_forwarded_per_node_out, *host_new_message, *device_new_message;
	struct RouterBuffer *host_router_buffer, *device_router_buffer;
	struct MessageBuffer *host_out_app, *host_in_app, *device_out_app,
			*device_in_app;
	dim3 threads, grid;
	int *device_traffic_table, *host_traffic_table;
	int node_number = partition.node_number;
	int offset = partition.offset;
	int total_node_number = partition.total_node_number;
	int ext_conn_nb = partition.number_of_external_connections;
	int commRank;
	
	
	Simulation_Parameters *device_simulation_parameters;
	Simulation_Parameters simulation_custom_parameters;

	// variables for MPI	
	int ierr;
	ierr = MPI_Comm_rank(MPI_COMM_WORLD, &commRank);
	struct Data_Flow_Unit data;
	int masters_number = partition.masters_number;
	MPI_Status status;	 
	
	float area_x = partition.area_x;
	float area_y = partition.area_y;
	float area_z = partition.area_z;
	int step_x, step_y, step_z;

	
	// These are values that are common to all of the masters. We don't need to change them
	int nb_tours = simulation_parameters.simulation_config.simulation_time;
	int cell_size = simulation_parameters.topology_config.area.geo_cell.cell_size_m;

	float m_send = simulation_parameters.environment_config.m_send;
	float m_recv = simulation_parameters.environment_config.m_recv;
	float b_send = simulation_parameters.environment_config.b_send;
	float b_recv = simulation_parameters.environment_config.b_recv;
	
	/* This part is used to specify particular area parameters for each master depending on data read from "partition" */
	/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
	/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
	
	simulation_custom_parameters = simulation_parameters;
	
	simulation_custom_parameters.topology_config.area.geo_cell.step_x = (int)((area_x * 1000) / cell_size);
	step_x = simulation_custom_parameters.topology_config.area.geo_cell.step_x;
	if (((int)(area_x * 1000)) % cell_size != 0) {
		simulation_custom_parameters.topology_config.area.geo_cell.step_x += 1;
		step_x = simulation_custom_parameters.topology_config.area.geo_cell.step_x;
		area_x = ((float)(step_x * cell_size))/1000.0;
	}
	simulation_custom_parameters.topology_config.area.x_km = area_x;
		
	simulation_custom_parameters.topology_config.area.geo_cell.step_y = (int)((area_y * 1000) / cell_size);
	step_y = simulation_custom_parameters.topology_config.area.geo_cell.step_y;
	if (((int)(area_y * 1000)) % cell_size != 0) {
		simulation_custom_parameters.topology_config.area.geo_cell.step_y += 1;
		step_y = simulation_custom_parameters.topology_config.area.geo_cell.step_y;
		area_y = ((float)(step_y * cell_size))/1000.0;
	}
	simulation_custom_parameters.topology_config.area.y_km = area_y;

	simulation_custom_parameters.topology_config.area.geo_cell.step_z = (int)((area_z * 1000) / cell_size);
	step_z = simulation_custom_parameters.topology_config.area.geo_cell.step_z;
	if (((int)(area_z * 1000)) % cell_size != 0) {
		simulation_custom_parameters.topology_config.area.geo_cell.step_z += 1;
		step_z = simulation_custom_parameters.topology_config.area.geo_cell.step_z;
		area_z = ((float)(step_z * cell_size))/1000.0;
	}
	simulation_custom_parameters.topology_config.area.z_km = area_z;
		
	simulation_custom_parameters.topology_config.area.geo_cell.cell_number = step_x * step_y; 
	if (simulation_parameters.simulation_config._3D_is_activated)
		simulation_custom_parameters.topology_config.area.geo_cell.cell_number *= step_z;
	int cell_number = simulation_custom_parameters.topology_config.area.geo_cell.cell_number;
	
	//printf("[Proc %d] - cell_nb %d, area_x %f, area_y %f, area_z %f\n", commRank, cell_number, area_x, area_y, area_z);
	
	/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
	
	//Get back MPI types
	MPI_Datatype mpi_master_buff = mpi_types.mpi_master_buff;
	MPI_Datatype mpi_data_flow = mpi_types.mpi_data_flow;
	
	hiprandGenerator_t *gen = (hiprandGenerator_t *)genV;
	
	grid.x = 32;

	/* Test of the node number to decide of the number and dimension of thread blocks */
	if (node_number % 32 == 0 && node_number > 0) {

		threads.x = node_number / 32;
	} else {
		threads.x = (node_number / 32) + 1;
	}

#ifdef __timing__
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
#endif	

	/* Will contain different probabilities that are going to be used for the routing of the messages */
	float *device_router_prob, *host_router_prob;

	hipMalloc((void**) &device_simulation_parameters, sizeof(Simulation_Parameters));
	checkCUDAError("cudaDeviceSimParamMalloc");

	hipMemcpy(device_simulation_parameters, &simulation_custom_parameters, sizeof(Simulation_Parameters),
			hipMemcpyHostToDevice);

	/* These parameters will allow us to analyze some features (comment to be changed) */
	hipHostMalloc((void**) &host_total_dest, sizeof(int));
	checkCUDAError("cudaBufferInMalloc");

	hipHostMalloc((void**) &host_new_message, node_number * sizeof(int));
	checkCUDAError("cudaNewMessageMalloc");

	hipHostMalloc((void**) &host_forwarded_per_node, node_number * sizeof(int));
	checkCUDAError("cudaHostForwardedPerNodeOutMalloc");

	hipHostMalloc((void**) &host_forwarded_per_node_out, node_number * sizeof(int));
	checkCUDAError("cudaHostForwardedPerNodeOutMalloc");

	hipMalloc((void**) &device_total_dest, sizeof(int));
	checkCUDAError("cudaDeviceTotalDestMalloc");

	hipMalloc((void**) &device_forwarded_per_node, node_number * sizeof(int));
	checkCUDAError("cudaDeviceForwardedPerNodeMalloc");

	hipMalloc((void**) &device_forwarded_per_node_out, node_number * sizeof(int));
	checkCUDAError("cudaDeviceForwardedPerNodeOutMalloc");

	hipMalloc((void**) &device_new_message, node_number * sizeof(int));
	checkCUDAError("cudaDeviceNewMessageMalloc");

	/* Initialization of the parameters */
	*host_total_dest = 0;

	hipMemcpy(device_total_dest, host_total_dest, sizeof(int),
			hipMemcpyHostToDevice);

	for (int i = 0; i < node_number; i++) {
		host_forwarded_per_node[i] = 0;
		host_forwarded_per_node_out[i] = 0;
		host_new_message[i] = -1;
	}
	hipMemcpy(device_forwarded_per_node, host_forwarded_per_node,
			node_number * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(device_forwarded_per_node_out, host_forwarded_per_node_out,
			node_number * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(device_new_message, host_new_message,
			node_number * sizeof(int), hipMemcpyHostToDevice);

	/* Receivers Host memory allocation */
	hipHostMalloc((void**) &host_traffic_table, node_number * sizeof(int));
	checkCUDAError("cudaRecMalloc");

	/* Receivers Device memory allocation */
	hipMalloc((void**) &device_traffic_table, node_number * sizeof(int));
	checkCUDAError("cudaDeviceRecMalloc");

	/* Cell host memory allocation */
	hipHostMalloc((void**) &host_cell, cell_number * sizeof(struct Cell));
	checkCUDAError("cudaCellMalloc");

	/* Cell Device memory allocation */
 	hipMalloc((void**) &device_cell, cell_number * sizeof(struct Cell));
 	checkCUDAError("cudaDeviceCellMalloc");

	/* Random position and speed parameters (needed for Geo) host memory allocation */
 	hipHostMalloc((void **) &host_pos_randx, node_number * sizeof(float));
 	checkCUDAError("cudaPosRandxMalloc");
 
	hipHostMalloc((void **) &host_pos_randy, node_number * sizeof(float));
	checkCUDAError("cudaPosRandyMalloc");
 
	hipHostMalloc((void **) &host_pos_randz, node_number * sizeof(float));
	checkCUDAError("cudaPosRandzMalloc");
 
	hipHostMalloc((void **) &host_v_randx, node_number * sizeof(float));
 	checkCUDAError("cudaVRandxMalloc");
 
	hipHostMalloc((void **) &host_v_randy, node_number * sizeof(float));
 	checkCUDAError("cudaVRandyMalloc");
 
	hipHostMalloc((void **) &host_v_randz, node_number * sizeof(float));
 	checkCUDAError("cudaVRandzMalloc");

	/* Random speed parameters (needed for Mobility) Device memory allocation */
 	hipMalloc((void **) &device_randx, node_number * sizeof(float));
 	checkCUDAError("cudaDeviceRandxMalloc");
 
	hipMalloc((void **) &device_randy, node_number * sizeof(float));
 	checkCUDAError("cudaDeviceRandyMalloc");
 
	hipMalloc((void **) &device_randz, node_number * sizeof(float));
 	checkCUDAError("cudaDeviceRandzMalloc");
 
	hipMalloc((void **) &device_randv, node_number * sizeof(float));
 	checkCUDAError("cudaDeviceRandvMalloc");
	
	/* Geo host memory allocation */
	hipHostMalloc((void**) &host_geo, node_number * sizeof(struct Geo));
	checkCUDAError("cudaGeoMalloc");

	/* Geo Device memory allocation */
	hipMalloc((void**) &device_geo, node_number * sizeof(struct Geo));
	checkCUDAError("cudaDeviceGeoMalloc");

	/* Host Buffer allocation */
	hipHostMalloc((void**) &host_in_app, node_number * sizeof(struct MessageBuffer));
	checkCUDAError("cudaBufferInMalloc");

	hipHostMalloc((void**) &host_out_app, node_number * sizeof(struct MessageBuffer));
	checkCUDAError("cudaBufferOutMalloc");

	hipHostMalloc((void**) &host_in_phy, node_number * sizeof(struct Buffer));
	checkCUDAError("cudaBufferInMalloc");

	hipHostMalloc((void**) &host_out_phy, node_number * sizeof(struct Buffer));
	checkCUDAError("cudaBufferOutMalloc");

	/* Device Buffer memory allocation */
	hipMalloc((void**) &device_in_phy, node_number * sizeof(struct Buffer));
	checkCUDAError("cudaDeviceBufferInMalloc");

	hipMalloc((void**) &device_out_phy, node_number * sizeof(struct Buffer));
	checkCUDAError("cudaDeviceOutPhyMalloc");
	
	hipMalloc((void**) &device_ext_connections, ext_conn_nb * sizeof(struct Connection));
	hipMemcpy(device_ext_connections, partition.external_connections, ext_conn_nb * sizeof(struct Connection),
			hipMemcpyHostToDevice);

	hipHostMalloc((void**) &host_remote_comm_out, ext_conn_nb * sizeof(struct RemoteCommBuffer));
	checkCUDAError("cudaRemCommOutMalloc");
	
	hipHostMalloc((void**) &host_remote_comm_in, ext_conn_nb * sizeof(struct RemoteCommBuffer));
	checkCUDAError("cudaRemCommInMalloc");

	/* Device Buffer memory allocation */
	hipMalloc((void**) &device_remote_comm_out, ext_conn_nb * sizeof(struct RemoteCommBuffer));
	checkCUDAError("cudaDeviceRemCommOutMalloc");
	
	hipMalloc((void**) &device_remote_comm_in, ext_conn_nb * sizeof(struct RemoteCommBuffer));
	checkCUDAError("cudaDeviceRemCommInMalloc");

	hipMalloc((void**) &device_in_app, node_number * sizeof(struct MessageBuffer));
	checkCUDAError("cudaDeviceInPhyMalloc");

	hipMalloc((void**) &device_out_app, node_number * sizeof(struct MessageBuffer));
	checkCUDAError("cudaDeviceOutAppMalloc");

	hipHostMalloc((void **) &host_router_buffer,
			node_number * sizeof(struct RouterBuffer));
	checkCUDAError("cudaRouterBufferMalloc");

	hipMalloc((void**) &device_router_buffer, node_number * sizeof(struct RouterBuffer));
	checkCUDAError("cudaDeviceRouterBufferMalloc");

	hipMalloc((void **) &device_router_prob, node_number * sizeof(float));
	checkCUDAError("cudaDeviceBufferProbMalloc");

	hipHostMalloc((void **) &host_router_prob, node_number * sizeof(float));
	checkCUDAError("cudaBufferProbMalloc");
	
	/* These buffers are used for the inter-master communication */
	/* Instead of sending the remote messages in the order in which we read them, we prefer to regroup them by master_addressee id */
	/* Here we allocate memory for both emission and reception of remote messages and initialize the e */
	master_in = (MasterBuffer *) malloc (masters_number * sizeof(MasterBuffer));
	master_out = (MasterBuffer *) malloc (masters_number * sizeof(MasterBuffer));
	for (int master = 0; master < masters_number; master++) {
		master_in[master].read_index = -1;
		master_in[master].write_index = 0;
		master_out[master].read_index = -1;
		master_out[master].write_index = 0;
		for (int elt = 0; elt < 2 * Maxelement; elt ++) {
			master_in[master].element[elt].header[0] = -1;
			master_in[master].element[elt].header[1] = -1;
			master_in[master].element[elt].header[2] = -1;
			master_in[master].element[elt].header[3] = -1;
			master_out[master].element[elt].header[0] = -1;
			master_out[master].element[elt].header[1] = -1;
			master_out[master].element[elt].header[2] = -1;
			master_out[master].element[elt].header[3] = -1;
		}
	}
	
	/* Host initialization */
	Init_host_random(host_cell, host_geo, host_pos_randx, host_pos_randy,
 	host_pos_randz, host_v_randx, host_v_randy, host_v_randz, node_number, simulation_custom_parameters,
 	offset, partition.external_connections, partition.number_of_external_connections);

	/****************************/
	/* Testing Geo values 

	printf("N = %d\n",N);
	for (int i = 0; i < N; i++){
		printf("Node = %d - PosX = %d - PosY = %d - PosZ = %d - SpeedX = %d - SpeedY = %d - SpeedZ = %d - CellID = %d\n",i,host_geo[i].P.x,host_geo[i].P.y,host_geo[i].P.z,host_geo[i].Speedx,host_geo[i].Speedy,host_geo[i].Speedz,host_geo[i].CellId);
	} */

	/* Device initialization */
	Init_device_random(host_cell, host_geo, device_cell, device_geo, node_number, simulation_custom_parameters);

	/*
	 * Setting of the different senders and receivers. Here the unique sender is 0 and the unique receiver is N-1
	 * Later, this may be replaced by a function. It depends on the scenario we want to apply
	 */


	/* May be doing it directly in the device could be better for the performance */
	if (simulation_parameters.application_config.predefined_traffic.application_type == DEFAULT){
		
		if (offset == 0)
			host_traffic_table[0] = total_node_number-1;
		else
			host_traffic_table[0] = -1;
		
		for (int i = 1; i < node_number; i++) {
			host_traffic_table[i] = -1;
		}
	}

	hipMemcpy(device_traffic_table, host_traffic_table, node_number * sizeof(int),
			hipMemcpyHostToDevice);

	/* Device buffers initialization */
	Init_Buffer<<<threads,grid>>>(device_out_phy, node_number);
	checkCUDAError("cuda kernel error Init OutBuff");

	Init_Buffer<<<threads,grid>>>(device_in_phy, node_number);
	checkCUDAError("cuda kernel error Init InBuff");

	//Initialization of remote comm buffers in the CPU since the number of these buffers is low
	for (int external_connection = 0; external_connection < ext_conn_nb; external_connection ++) {
		host_remote_comm_out[external_connection].write_index = 0;
		host_remote_comm_out[external_connection].read_index = -1;
		host_remote_comm_out[external_connection].conn_node_id = partition.external_connections[external_connection].node1;
		host_remote_comm_in[external_connection].write_index = 0;
		host_remote_comm_in[external_connection].read_index = -1;
		host_remote_comm_in[external_connection].conn_node_id = partition.external_connections[external_connection].node1;
		for (int i = 0; i < Maxelement; i++) {
			host_remote_comm_out[external_connection].element[i].header[0] = -1;
			host_remote_comm_out[external_connection].element[i].header[1] = -1;
			host_remote_comm_out[external_connection].element[i].header[2] = -1;
			host_remote_comm_out[external_connection].element[i].header[3] = -1;
			host_remote_comm_in[external_connection].element[i].header[0] = -1;
			host_remote_comm_in[external_connection].element[i].header[1] = -1;
			host_remote_comm_in[external_connection].element[i].header[2] = -1;
			host_remote_comm_in[external_connection].element[i].header[3] = -1;
		}
	}
	
	//Then copy to the GPU
	hipMemcpy(device_remote_comm_out, host_remote_comm_out, ext_conn_nb * sizeof(struct RemoteCommBuffer),
			hipMemcpyHostToDevice);
	hipMemcpy(device_remote_comm_in, host_remote_comm_in, ext_conn_nb * sizeof(struct RemoteCommBuffer),
			hipMemcpyHostToDevice);
			

	Init_App_Buffer<<<threads,grid>>>(device_in_app, node_number);
	checkCUDAError("cuda kernel error Init InAppBuff");

	Init_App_Buffer<<<threads,grid>>>(device_out_app, node_number);
	checkCUDAError("cuda kernel error Init OutAppBuff");

	/* This may only be useful for debugging */
	//hipMemcpy(host_out_phy,device_out_phy, N * sizeof(struct Buffer), hipMemcpyDeviceToHost);

	/* When are we going to use it ? */
	Init_Router<<<threads,grid>>>(device_router_buffer, node_number);

	// ?? Only with memcheck ? why ?
	checkCUDAError("cuda kernel error init router");

	/* Initialization of the random number generator */
	hiprandSetPseudoRandomGeneratorSeed(*gen, time(NULL));
	
	/* Setting of the random initial positions and the random speed */ 	
	hiprandGenerateUniform ( *gen , device_randx , node_number );
	hiprandGenerateUniform ( *gen , device_randy , node_number );
	hiprandGenerateUniform ( *gen , device_randz , node_number );
 	hiprandGenerateUniform ( *gen , device_randv , node_number );

	//////////////////////////
	// HERE SCHEDULING STEP //
	//////////////////////////

	Schedule (0, 1, (nb_tours + partition.additional_time) / 5, DISTRIB, 1, MOB, nb_tours + partition.additional_time);
	Schedule (0, 2, (nb_tours + partition.additional_time) / 5, DISTRIB, 1, CON, nb_tours + partition.additional_time);
	//To be scheduled before PROTO_OUT absolutely!
	//Schedule (0, 4, 215, DISTRIB, 1, TC_OUT, nb_tours+100);
	//The last APP-level called kernel that is the one which message is kept because for the moment the two APP kernels share the same
	//message buffer and always write their message at the first place

	Schedule (0, 3, nb_tours + partition.additional_time, CONSEC, 1, APP_OUT, nb_tours + partition.additional_time);
	Schedule (0, 5, nb_tours + partition.additional_time, CONSEC, 1, PROTO_OUT, nb_tours + partition.additional_time);
	Schedule (0, 6, nb_tours + partition.additional_time, CONSEC, 1, PKT_OUT, nb_tours + partition.additional_time);
	Schedule (0, 7, nb_tours + partition.additional_time, CONSEC, 1, PKT_IN, nb_tours + partition.additional_time);
	Schedule (0, 8, nb_tours + partition.additional_time, CONSEC, 1, PROTO_IN, nb_tours + partition.additional_time);
	Schedule (0, 9, nb_tours + partition.additional_time, CONSEC, 1, APP_IN, nb_tours + partition.additional_time);

	Calculate_timestamps(simulation_custom_parameters);
	//Print_event_list();

	//////////////////////////

	for (int i = 0; i < nb_tours + partition.additional_time; i++) {

		//printf("[Master %d] Round %d\n", commRank, i);
		
		struct Event event;
		hiprandGenerateUniform ( *gen , device_router_prob , node_number );
		
		// If we have to communicate with a GUI
		if (gui) {
			
			Data_Flow_Unit data;
			if (i>0){
				hipMemcpy(host_geo, device_geo, node_number * sizeof(struct Geo),hipMemcpyDeviceToHost);
				hipMemcpy(host_new_message, device_new_message, node_number * sizeof(int),hipMemcpyDeviceToHost);
			}
			for (int node = 0; node < node_number; node++) {
				data.geo[node].neighbor_number = host_geo[node].neighbor_number;
				data.geo[node].p = host_geo[node].p;
				data.new_message[node] = host_new_message[node];
				for(int j = 0; j<host_geo[node].neighbor_number; j++)
                        		data.geo[node].neighbor_vector[j] = host_geo[node].neighbor_vector[j];
			}
			
			data.tour = i;
			data.nb_node = node_number;
			
			ierr = MPI_Send ( &data, 1, mpi_data_flow, 0, 0, MPI_COMM_WORLD ) ;
		} 

#ifdef __timing__		
		hipEventRecord(start1, 0);
#endif
			
		while (Next_event_of_the_round (&event, i)){
			//if (commRank == 2) printf("%d is at round %d doing %d\n", commRank, i, event.type);
			switch (event.type){
			
				case MOB:
				for (int j=0; j < event.frequency; j++){
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif

					Mobility<<<threads,grid>>>(device_geo, device_randx, device_randy, device_randz, device_randv, node_number, device_simulation_parameters);

#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][0], start2, stop2);
#endif	
					
					checkCUDAError("cuda kernel error Mob");

#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					Mobility_Control<<<threads,grid>>>(device_cell,device_geo, node_number, device_simulation_parameters);	

#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][1], start2, stop2);
#endif
					checkCUDAError("cuda kernel error Mob ctrl");
				}
				break;
				
				case CON:
				for (int j=0; j < event.frequency; j++){
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					Update_Cell<<<(cell_number % 32 == 0 ? cell_number/32 : cell_number/32 + 1),32>>>(cell_number, device_cell, node_number);
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][2], start2, stop2);
#endif
					checkCUDAError("cuda kernel error Up cell");
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					Visible_Opt<<<threads,grid>>>(device_geo, device_cell,node_number,cell_size,
						offset, device_ext_connections, partition.number_of_external_connections);
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][3], start2, stop2);
#endif
					checkCUDAError("cuda kernel error Visib");
				/*	
					hipMemcpy(host_geo, device_geo, node_number * sizeof(struct Geo),hipMemcpyDeviceToHost);
					if (commRank == 1)
						for (int ne = 0; ne < host_geo[0].neighbor_number; ne++) {
							printf("[rd = %d] nb = %d N: %d\n", i, host_geo[0].neighbor_number, host_geo[0].neighbor_vector[ne]);
						}
						
					
					/* 
			 		 * If we would like to test the connectivity step 
			 		 * hipMemcpy(host_geo, device_geo, node_number * sizeof(struct Geo),hipMemcpyDeviceToHost);
			 		 * hipMemcpy(host_cell, device_cell, cell_number * sizeof(struct Cell),hipMemcpyDeviceToHost);
			 		 * Connectivity_control(host_geo, node_number, i, host_cell, cell_size);
			 		 */	 
				}
				break;
				
				case APP_OUT:

				for (int j=0; j < event.frequency; j++) {
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					Application_Out<<<threads,grid>>>(device_out_app, device_traffic_table, 3, node_number, i*f+j, nb_tours, offset);
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][4], start2, stop2);
#endif
					checkCUDAError("cuda kernel error App out");
				}
				break;
				
				case APP_IN:
				for (int j=0; j < event.frequency; j++) {
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					Message_In<<<threads,grid>>>(device_in_app, node_number, device_total_dest);
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][5], start2, stop2);
#endif
					checkCUDAError("cuda kernel error Msg In");
				}
				break;
				
				case PROTO_OUT:
				for (int j=0; j < event.frequency; j++){
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					Router_Out<<<threads,grid>>>(device_out_phy, device_out_app, i*f+j, node_number, offset);
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][6], start2, stop2);
#endif
					checkCUDAError("cuda kernel error Router out");
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					Reset_Buffer<<<threads,grid>>>(device_in_phy, node_number);
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][7], start2, stop2);
#endif
					checkCUDAError("cuda kernel error Reset Buffer");
				}
				break;
				
				case PROTO_IN:
				for (int j=0; j < event.frequency; j++) {
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					Router_In<<<threads,grid>>>(device_in_phy,device_out_phy,device_in_app,device_router_buffer,device_router_prob, drop_prob, node_number, offset);
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][8], start2, stop2);
#endif
					checkCUDAError("cuda kernel error Router In");
				}
				break;
				
				case PKT_OUT:
				for (int j=0; j < event.frequency; j++) {

					
					for (int ext_conn = 0; ext_conn < ext_conn_nb; ext_conn++) {
						host_remote_comm_out[ext_conn].write_index = 0;
					}
			
					hipMemcpy(device_remote_comm_out, host_remote_comm_out, ext_conn_nb * sizeof(struct RemoteCommBuffer), hipMemcpyHostToDevice);

#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					
					Sender<<<threads,grid>>>(device_geo,device_in_phy,device_out_phy, node_number, device_forwarded_per_node, device_forwarded_per_node_out, device_new_message, offset, device_remote_comm_out, m_send, b_send);
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][9], start2, stop2);
#endif
					checkCUDAError("cuda kernel error Sender");

					hipMemcpy(host_forwarded_per_node, device_forwarded_per_node, node_number * sizeof(int), hipMemcpyDeviceToHost);
					hipMemcpy(host_forwarded_per_node_out, device_forwarded_per_node_out, node_number * sizeof(int), hipMemcpyDeviceToHost);
					kernel_duration[i][15] = 0.0;
					kernel_duration[i][16] = 0.0;
					for (int node=0; node<node_number; node++) {
						//printf("Node = %d ==> frwd = %d", node, host_forwarded_per_node[node]);
						kernel_duration[i][15] += (float)host_forwarded_per_node[node];
						kernel_duration[i][16] += (float)host_forwarded_per_node_out[node];
					}

#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					hipMemcpy(host_remote_comm_out, device_remote_comm_out, ext_conn_nb * sizeof(struct RemoteCommBuffer), hipMemcpyDeviceToHost);
					
					for (int master = 0; master < masters_number; master++) {
						master_out[master].write_index = 0;
					}
			
			
					/* This loop organizes the remote messages in the "masters" local buffers before beginning the 
			   		effective communication with the other masters */   
					//printf("\nRound[%d] - Master = %d - ", i, commRank);
					for (int ext_conn = 0; ext_conn < ext_conn_nb; ext_conn++) {
					//printf("ext_conn = %d - remote_msgs = %d for: ", partition.external_connections[ext_conn].node1, host_remote_comm_out[ext_conn].write_index);
						if (host_remote_comm_out[ext_conn].write_index > 0) {
							for (int remote_msg = 0; remote_msg < host_remote_comm_out[ext_conn].write_index; remote_msg++) {
								int master_addressee = 0;
								int worker_addressee = host_remote_comm_out[ext_conn].ext_conn[remote_msg];
						
								// Looking for the master responsible of the worker addressee
								while (partition.conversion_table[master_addressee] < worker_addressee && master_addressee < masters_number - 1)
									master_addressee++;
								if (partition.conversion_table[master_addressee] > worker_addressee)
									master_addressee--;
							
								//Writing the message in the master's buffer if there is some place...
								if (master_out[master_addressee].write_index < (2 * Maxelement) - 1) {
									int index = master_out[master_addressee].write_index;
									struct Element elt = host_remote_comm_out[ext_conn].element[remote_msg];
							
									//printf("(mast: %d, worker: %d) ",master_addressee + 1, worker_addressee);
									master_out[master_addressee].ext_conn[index] = host_remote_comm_out[ext_conn].ext_conn[remote_msg];
									master_out[master_addressee].element[index] = elt;
									master_out[master_addressee].write_index++;
								}
							}
						}
					}
			
			
					/* Begginning of the real communication */
			
					//Sending step 	
					for (int master = 0; master < masters_number; master++) {
						if (master_out[master].write_index > 0) {
				
							int msg_nb = master_out[master].write_index;
							//TAG 0 ==> there are messages
							ierr = MPI_Send( &master_out[master], 1, mpi_master_buff, master + 1, 0, MPI_COMM_WORLD ) ;
							//ierr = MPI_Send( master_out[master].element, 1, mpi_element, master + 1, 0, MPI_COMM_WORLD ) ; 
		
						} else {
							//TAG 1 ==> No messages
							ierr = MPI_Send( &master_out[master], 1, mpi_master_buff, master + 1, 1, MPI_COMM_WORLD ) ; 
						}
					}
				
					for (int ext_conn = 0; ext_conn < ext_conn_nb; ext_conn++)
						host_remote_comm_in[ext_conn].write_index = 0;
		
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][10], start2, stop2);
#endif

#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
		
					//Receiving step				
					for (int master = 0; master < masters_number; master++) {
					
						ierr = MPI_Recv(&master_in[master], 1, mpi_master_buff, MPI_ANY_SOURCE, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
						if (status.MPI_TAG == 0) {
							
							//printf("[Round %d] Process %d received %d messages from process %d: the %dth message \n", i, commRank, master_in[master].write_index, status.MPI_SOURCE, master_in[master].element[0].header[2]);
						
							for (int msg = 0; msg < master_in[master].write_index; msg++) {
								int node = master_in[master].ext_conn[msg];
								int ext_conn = 0;
							
								//if (commRank == 2 && i % 20 == 0) printf("node = %d ",node);
							
								while (host_remote_comm_in[ext_conn].conn_node_id != node && ext_conn < partition.number_of_external_connections)
									ext_conn++;
							
								//if (commRank == 2 && i % 20 == 0) printf("ext_conn %d\n",ext_conn);
							
								if (host_remote_comm_in[ext_conn].write_index < Maxelement) {
									host_remote_comm_in[ext_conn].element[host_remote_comm_in[ext_conn].write_index] = master_in[master].element[msg];
									host_remote_comm_in[ext_conn].write_index++;
								}
							}
						
						} //else if (commRank == 2 && i % 20 == 0) printf("[Round %d] Process %d received nothing from process %d\n", i, commRank, status.MPI_SOURCE);
					}
				
					//printf("Before -- Write Index = %d\n",host_remote_comm_in[0].write_index);
					hipMemcpy(device_remote_comm_in, host_remote_comm_in, ext_conn_nb*sizeof(struct RemoteCommBuffer), hipMemcpyHostToDevice);
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][11], start2, stop2);
#endif
					
				}
				//hipMemcpy(host_geo, device_geo, node_number * sizeof(struct Geo), hipMemcpyDeviceToHost);
					//for (int node = 0; node < node_number; node++)
						//printf("Node 0: Energy level = %f with %d neighbors after Send of round %d\n", host_geo[0].energy, host_geo[0].neighbor_number, i);
				break;
				
				case PKT_IN:
				for (int j=0; j < event.frequency; j++){
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					//data[0] = i;
					//ierr = MPI_Send ( data, 2, MPI_INT, 0, 0, MPI_COMM_WORLD ) ;
					// Synchronization point. Before receiving any message, we make sure all processes reached this 
					// step, which means that whatever node intending to send a message in this round has finished 
					/* This would be more coherent if we transform all the 6 event - block into a comm_event ? */
					// This seems to make nodes rather too dependent ! if a node sends, the receiver is immediately
					// called to receive the message (the three levels PKT - PROTO - APP)
					// is it tolerable ?

					ierr = MPI_Barrier ( comm_masters );
#ifdef __timing__
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&kernel_duration[i][12], start2, stop2);
#endif
#ifdef __timing__
			hipEventRecord(start2, 0);
#endif
					Receiver<<<threads,grid>>>(device_in_phy, node_number, device_remote_comm_in, device_geo, offset, m_recv, b_recv);
#ifdef __timing__
			float time; 
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&time, start2, stop2);
			kernel_duration[i][13] += time;
#endif
					checkCUDAError("cuda kernel error Receiver");
				}
				break;
				
				case TC_OUT:
					TC_Out<<<threads,grid>>>(device_out_app, device_traffic_table, node_number, i*f, nb_tours);
					checkCUDAError("cuda kernel error tc out");
				break;
				
			}//switch
			
			/* This is for the validation of the timestamp ==> OK
			switch (event.type){
				case MOB:
				printf("MOB: freq = %d - ts = %f\n",event.frequency, event.timestamp);
				break;
				case CON:
				printf("CON: freq = %d - ts = %f\n",event.frequency, event.timestamp);
				break;
				case APP_OUT:
				printf("APP_OUT: freq = %d - ts = %f\n",event.frequency, event.timestamp);
				break;
				case APP_IN:
				printf("APP_IN: freq = %d - ts = %f\n",event.frequency, event.timestamp);
				break;				
				case PROTO_OUT:
				printf("PROTO_OUT: freq = %d - ts = %f\n",event.frequency, event.timestamp);
				break;
				case PROTO_IN:
				printf("PROTO_IN: freq = %d - ts = %f\n",event.frequency, event.timestamp);
				break;
				case PKT_OUT:
				printf("PKT_OUT: freq = %d - ts = %f\n",event.frequency, event.timestamp);
				break;
				case PKT_IN:
				printf("PKT_IN: freq = %d - ts = %f\n",event.frequency, event.timestamp);
				break;
			}
			*/

			
			
			
		}//while
#ifdef __timing__	

			hipEventRecord(stop1, 0);
			hipEventSynchronize(stop1);
			hipEventElapsedTime(&kernel_duration[i][14], start1, stop1);

#endif
		
	}//for

	ierr = MPI_Send( &data, 1, mpi_data_flow, 0, 1, MPI_COMM_WORLD ) ;

#ifdef __timing__	

	//Database: creation and connection
	conn = mysql_init (NULL);
			
	if (mysql_real_connect (conn,host_name,user_name,password,db_name,port_num,socket_name,flags) == NULL){
  		fprintf(stderr, "Connection to database failed, error : %s\n", mysql_error(conn));
  	}

	for (int round = 0; round < 1000; round ++) {
		sprintf(query,"INSERT INTO %d_128_1_1_8 (round, master, round_len, MOB, CON, APP_OUT, APP_IN, PROTO_OUT, PROTO_IN, SND_M, SND_K, RCV_M, RCV_K, SYNC, FRWD_PKT, FRWD_PKT_OUT)", total_node_number);
		sprintf(query, "%s VALUES ('%d', '%d', '%f', '%f', '%f', '%f', '%f', '%f', '%f', '%f', '%f', '%f', '%f', '%f', '%d', '%d')",query,round,commRank,
			kernel_duration[round][14], kernel_duration[round][0] + kernel_duration[round][1], kernel_duration[round][2] +
			kernel_duration[round][3], kernel_duration[round][4], kernel_duration[round][5], kernel_duration[round][6] +
			kernel_duration[round][7], kernel_duration[round][8], kernel_duration[round][10], kernel_duration[round][9],
			kernel_duration[round][11], kernel_duration[round][13], kernel_duration[round][12], (int) kernel_duration[round][15],
			(int) kernel_duration[round][16]); 
		
		process_query (conn,query);
	}

	mysql_close (conn);
#endif
	
	hipMemcpy(host_total_dest, device_total_dest, sizeof(int) , hipMemcpyDeviceToHost);
	hipMemcpy(host_forwarded_per_node, device_forwarded_per_node, node_number * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(host_forwarded_per_node_out, device_forwarded_per_node_out, node_number * sizeof(int), hipMemcpyDeviceToHost);

	int absolute_total_forwarded = 0;

#ifdef __timing__
	for (int i=0; i<nb_tours + partition.additional_time; i++)
		absolute_total_forwarded += (int) kernel_duration[i][15] + (int) kernel_duration[i][16];
#else
	for (int i=0; i<node_number; i++) 
		absolute_total_forwarded += host_forwarded_per_node[i] + host_forwarded_per_node_out[i];

#endif

	performance.total_dest = *host_total_dest;
	performance.total_forwarded = absolute_total_forwarded;
	
	performance.host_used_memory =
		(float) node_number * 
		      (float) (	3 * sizeof(int) + 
			sizeof(struct Geo) + 
			2 * sizeof(struct Buffer) + 
			2 * sizeof(struct MessageBuffer) +
			2 * sizeof(struct RemoteCommBuffer) +
			7 * sizeof(float) + 
			sizeof(RouterBuffer)  ) +
			(float) cell_number * (float) sizeof(struct Cell) + 
			(float) sizeof(int);

	performance.device_used_memory =
		(float) node_number * 
		      (float) (	3 * sizeof(int) + 
			sizeof(struct Geo) + 
			2 * sizeof(struct Buffer) + 
			2 * sizeof(struct MessageBuffer) +
			2 * sizeof(struct RemoteCommBuffer) +
			5 * sizeof(float) + 
			sizeof(RouterBuffer)  ) +
			(float) sizeof(int) +
			(float) cell_number * (float) sizeof(struct Cell) + 
			(float) ext_conn_nb * (float) sizeof(struct Connection) +
			(float) sizeof(Simulation_Parameters);
	

	/* Freeing the memory at the end of computing */

	hipFree(device_geo);
	hipFree(device_ext_connections);
	hipFree(device_remote_comm_out);
	hipFree(device_remote_comm_in);
	hipFree(device_new_message);
	hipFree(device_cell);
	hipFree(device_in_phy);
	hipFree(device_out_phy);
	hipFree(device_router_buffer);
	hipFree(device_router_prob);
	hipFree(device_in_app);
	hipFree(device_out_app);
	hipFree(device_traffic_table);
	hipFree(device_total_dest);
	hipFree(device_forwarded_per_node);
	hipFree(device_forwarded_per_node_out);
	hipFree(device_randx);
	hipFree(device_randy);
	hipFree(device_randz);
	hipFree(device_randv);
	hipFree(device_simulation_parameters);

	hipHostFree(host_geo);
	hipHostFree(host_remote_comm_out);
	hipHostFree(host_remote_comm_in);
	hipHostFree(host_new_message);
	hipHostFree(host_cell);
	hipHostFree(host_in_phy);
	hipHostFree(host_out_phy);
	hipHostFree(host_router_buffer);
	hipHostFree(host_router_prob);
	hipHostFree(host_in_app);
	hipHostFree(host_out_app);
	hipHostFree(host_traffic_table);
	hipHostFree(host_total_dest);
	hipHostFree(host_forwarded_per_node);
	hipHostFree(host_forwarded_per_node_out);
	hipHostFree(host_pos_randx);
	hipHostFree(host_pos_randy);	
	hipHostFree(host_pos_randz);
	hipHostFree(host_v_randx);
	hipHostFree(host_v_randy);	
	hipHostFree(host_v_randz);
	
	return performance;
}
