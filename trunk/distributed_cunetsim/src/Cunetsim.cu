#include "hip/hip_runtime.h"
/*******************************************************************************

  Eurecom Cunetsim2
  Copyright(c) 2011 - 2012 Eurecom

  This program is free software; you can redistribute it and/or modify it
  under the terms and conditions of the GNU General Public License,
  version 2, as published by the Free Software Foundation.

  This program is distributed in the hope it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for
  more details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc.,
  51 Franklin St - Fifth Floor, Boston, MA 02110-1301 USA.

  The full GNU General Public License is included in this distribution in
  the file called "COPYING".

  Contact Information
  Cunetsim Admin: cunetsim@eurecom.fr
  Cunetsim Tech : cunetsim_tech@eurecom.fr
  Forums       : TODO
  Address      : Eurecom, 2229, route des crêtes, 06560 Valbonne Sophia Antipolis, France

*******************************************************************************/

/**
 * \file Cunetsim.cu
 * \brief Program that launches the scenario, displays results and finishes the simulation
 * \author Bilel BR
 * \version 0.0.2
 * \date 
 */
 
 
// Host defines
#define NUM_THREADS 8
#define STR_SIZE 50

// Includes
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <fcntl.h>
#include <string.h>
#include <math.h>
#include <mpi.h>

#ifndef STRUCTURES_H_
#define STRUCTURES_H_
#include "structures.h"
#include "/usr/local/cuda/include/hiprand.h"
#endif /* STRUCTURES_H_ */
#ifndef INTERFACES_H_
#define INTERFACES_H_
#include "interfaces.h"
#endif /* INTERFACES_H_ */

#include "vars.h"

//#define _control_
//#define _printgraph_
//#define __timing2__

#define __timing__
#define __mobonly__

/* global structure */
extern Simulation_Parameters simulation_parameters;

float kernel_duration[1000][17];

void process_result_set (MYSQL *conn, MYSQL_RES *res_set){
  MYSQL_ROW	row;
  unsigned int	i;
  
  while ((row = mysql_fetch_row (res_set)) != NULL){
    mysql_field_seek (res_set, 0);
    
    for (i = 0; i < mysql_num_fields (res_set); i++){
      mysql_fetch_field (res_set);
      printf (" %s |",row[i]);
    }
    printf("\n");
  }
  printf ("%lu rows returned\n", (unsigned long) mysql_num_rows (res_set));
}


int process_query (MYSQL *conn, char *query){

  MYSQL_RES *res_set;
  
  if (mysql_real_query (conn, query, strlen(query)) != 0){
    printf("process_query() %s failed => error: %s\n", query, mysql_error(conn));
    return -1; //exit (-1);
  }
  
  res_set = mysql_store_result (conn);
  if (res_set != NULL){
    process_result_set (conn, res_set);
    mysql_free_result (res_set);
    return 0;
  }

  return -1;
}



void Print_help(){
	printf("\n*****************************************************************************************\n");
	printf("****************************************  HELP  *****************************************\n\n");
	printf("Usage: ./Cunetsim [-h] [-d device] [-g gui_pipe_descriptor]\n");
	printf ("-h provides this help message!\n");
  	printf ("-d [0-device_number] sets the current device\n");	
  	printf ("-g descriptor sets the file descriptor of the communication pipe\n(the communication is between the simulator and the GUI)\n");
  	printf("\n*****************************************************************************************\n");
}

 /**
 * \fn int main(int argc, char** argv)
 * \brief calls the wanted scenario function and gets back the monitoring data
 *
 * \param argc number of arguments passed to the program
 * \param argv vector containing the arguments passed to the program
 * \return 0 if all went well
 */
int main(int argc, char** argv) {
	
	hipEvent_t start, stop;// timers to compute the runtime
	hiprandGenerator_t gen;	// seed for the random generator
	float elapsed, total_elapsed = 0.0, total_dest = 0.0; //runtime monitoring var
	float elapsed_min = 10000000.0, elapsed_max = 0.0;
	float total_forwarded = 0.0; // traffic monitoring var
	float drop_probability; // 
	enum Initial_Distribution initial_distribution;
	struct Final_Data final_data;
	char output_format = 0;
	int commSize, commRank, masters_commRank;
	int ierr ;
	struct Data_Flow_Unit data, data_flow;
	struct MPI_data_types mpi_types;
	float final_result [7];
	MPI_Status status ;
	MPI_Group  group_world, group_masters;
	MPI_Comm   comm_masters, comm_world;
	int manager_id = 0;
	int number_of_executions = 1;
	char host[80];  
  	int length;
  	int offset, total_node_number;
  	float area_x, area_y, area_z;

#ifdef __timing__
	
	// Database variables
	MYSQL *conn = NULL;
	char *host_name = "pyroclaste";
  	char *user_name = "root";
  	char *password  = "database";
 	unsigned int port_num = 0;
  	char *socket_name = NULL;
  	char *db_name = "cunetsim";
  	int flags = 0;
	char query[4096];

#endif

	int node_number, nb_tours; //nb_tours is the number of rounds 
	struct Performances performance;
	struct Partition partition;
	int write_descriptor;
	
	//default values in case no options were given
	write_descriptor = -3;
	output_format = 0;
	char c;
	
	// Initialize MPI state
    	MPI_Init(NULL, NULL);
		
    	ierr = MPI_Comm_size(MPI_COMM_WORLD, &commSize);
 	ierr = MPI_Comm_rank(MPI_COMM_WORLD, &commRank);
	
	while ((c = getopt (argc, argv, "hgd:n:")) != -1) {
	 	switch (c) {
	 		case 'h':
	 		Print_help();
	 		exit(0);
	 		break;
	 		
	 		case 'g':
	 		if (commRank == 0)
	 			// Only the coordinator opens the communication tube with the GUI since both are running on the same machine
	 			write_descriptor = open("../../Debug/comm_tube", O_WRONLY);
	 		output_format = 1; // All the processes use this parameter to know whether the visualizor is on
	 		break;
	 		
	 		case 'd':
	 		hipSetDevice(atoi(optarg));
	 		break;
	 		
	 		case 'n':
	 		node_number = atoi(optarg);
	 		break;
	 		
	 		default:
	 		Print_help();
	 		exit (-1);
	 		break;
	 		
	 	}
	 
	}

	/* Initialization of the random number generator */
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	checkCUDAError("cuda create random");

	// Select the right GPU
	hipSetDevice((commRank <= 2 ? 0 : 1));

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	performance.total_dest = 0;
	
	for (int H = 1000; H <= 1000; (H < 65 ? H+= 2 : H += 10))
	//for (int H = 2; H <= 256; (H < 65 ? (H < 20 ? H++ : H+= 2) : H += 10))
	//for (int H = 356; H <= 446; (H < 65 ? (H < 20 ? H++ : H+= 2) : H += 10)) 
	{			

		simulation_parameters.simulation_config.node_number = H*H;
	
		/* Setting default values for the simulation parameters */
		Init_simulation_parameters();
		
		/* Getting back values that are necessary to launch the simulation */
		drop_probability = simulation_parameters.simulation_config.drop_probability;
		node_number = simulation_parameters.simulation_config.node_number;
		total_node_number = node_number;
		nb_tours = simulation_parameters.simulation_config.simulation_time;
		initial_distribution = simulation_parameters.topology_config.distribution.initial_distribution;

		total_elapsed = 0.0;
		total_dest = 0.0;
		elapsed_max = 0.0;
		elapsed_min = 10000000.0;
		total_forwarded = 0.0;

#ifdef __timing__
		
		for (int k=0; k<1000; k++)
			for (int l=0; l < 13; l++){
				kernel_duration[k][l] = 0.0;
			}
#endif 		

 	   	MPI_Get_processor_name(host, &length);  /* Get name of this processor */
 	 		
		/* Creating a new communicator excluding the manager (rank 0) for the synchronization */
		//Biiig error
		//if (commRank != 0) {

		comm_world = MPI_COMM_WORLD;

		/* Extract the original group handle */	  		
		MPI_Comm_group(comm_world, &group_world);
  		ierr = MPI_Group_excl(group_world, 1, &manager_id, &group_masters);  /* process 0 not member */
 		ierr = MPI_Comm_create(comm_world, group_masters, &comm_masters);	
		ierr = MPI_Group_rank (group_masters, &masters_commRank);
		
		// Step of MPI types creation //
		/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
	
		struct MasterBuffer master_buff;
		struct Element elt;
	
		// Variables needed to create a new MPI type which will encapsulate our 'struct Element'
		int blocklengths[MPI_STRUCT_ELTS + 11];
		MPI_Datatype types[MPI_STRUCT_ELTS + 11];
		MPI_Aint displacements[MPI_STRUCT_ELTS + 11];
	
		MPI_Datatype mpi_element, mpi_master_buff, mpi_position, mpi_geo, mpi_data_flow, mpi_connection, mpi_partition;
	
		MPI_Aint extent, element_extent, position_extent, geo_extent, connection_extent, float_extent;

		MPI_Type_extent(MPI_INT, &extent);
		MPI_Type_extent(MPI_FLOAT, &float_extent);

		types[0]= MPI_INT;
	   	types[1]= MPI_INT;
	    	types[2]= MPI_INT;

		blocklengths[0]= 1;         
	    	blocklengths[1]= 4;        
	    	blocklengths[2]= PAYLOAD_SIZE;
	    
	    	displacements[0]= 0;
	    	displacements[1]= extent;
	    	displacements[2]= 5 * extent;
   	
   		//Creation of the type mpi_element
   		ierr = MPI_Type_struct(MPI_STRUCT_ELTS, blocklengths, displacements, types, &mpi_element);
    		MPI_Type_commit(&mpi_element);

		//Preparation for the 2nd MPI data type creation, which is going to be transfered between masters
		types[0]= mpi_element;
    		types[1]= MPI_INT;
    		types[2]= MPI_INT;
    		types[3]= MPI_INT;

    		blocklengths[0]= 2 * Maxelement;         
    		blocklengths[1]= 1 * Maxelement;        
    		blocklengths[2]= 1;
    		blocklengths[3]= 1;
    
    		MPI_Type_extent(mpi_element, &element_extent);
    
    		displacements[0]= 0;
    		displacements[1]= 2 * Maxelement * element_extent;
    		displacements[2]= 2 * Maxelement * (extent + element_extent);
    		displacements[3]= 2 * Maxelement * (extent + element_extent) + extent;
   	
   		//Creation of the type mpi_master_buff
    		ierr = MPI_Type_struct(MPI_STRUCT_ELTS + 1, blocklengths, displacements, types, &mpi_master_buff);
    		MPI_Type_commit(&mpi_master_buff);
    
    		//Preparation of a 3rd mpi new type that contains a 3d coordinates ==> struct Position
    		types[0]= MPI_INT;
    		types[1]= MPI_INT;
    		types[2]= MPI_INT;
    
  		blocklengths[0]= 1;         
  	  	blocklengths[1]= 1;        
  	  	blocklengths[2]= 1;
    
    		displacements[0]= 0;
    		displacements[1]= extent;
    		displacements[2]= 2 * extent;
    
    		//Creation of the type mpi_position
    		ierr = MPI_Type_struct(MPI_STRUCT_ELTS, blocklengths, displacements, types, &mpi_position);
    		MPI_Type_commit(&mpi_position);
    
    		//Preparation of a 4th mpi new type that contains geo data for a node ==> struct Geo
    		MPI_Type_extent(mpi_position, &position_extent);
    
    		types[0] = mpi_position;
    		blocklengths[0]= 1;
    		displacements[0]= 0;

		types[13] = MPI_FLOAT;
    		blocklengths[13]= 1;
    		displacements[13]= position_extent + (11 + Maxneighbor) * extent;
    
    		for (int t = 1; t < 13; t++) {
    			types[t] = MPI_INT;
    			blocklengths[t]= 1;
    			displacements[t]= position_extent + (t - 1) * extent;
    		}
    
    		blocklengths[12] = Maxneighbor;
    
    		//Creation of the type mpi_geo
    		ierr = MPI_Type_struct(MPI_STRUCT_ELTS + 11, blocklengths, displacements, types, &mpi_geo);
    		MPI_Type_commit(&mpi_geo);
    
    		//Preparation of a 5th mpi new type that contains geo data for a number of nodes ==> struct Data_Flow_Unit
    		MPI_Type_extent(mpi_geo, &geo_extent);
    
    		types[0] = mpi_geo;
    		types[1] = MPI_INT;
    		types[2] = MPI_INT;
    		types[3] = MPI_INT;
    
    		blocklengths[0] = NB_DISPLAYED_NODES;
    		blocklengths[1] = NB_DISPLAYED_NODES;  
    		blocklengths[2] = 1;
    		blocklengths[3] = 1;
    
   	 	displacements[0] = 0;
   	 	displacements[1] = NB_DISPLAYED_NODES * geo_extent;
    		displacements[2] = NB_DISPLAYED_NODES * (extent + geo_extent);
    		displacements[3] = NB_DISPLAYED_NODES * (extent + geo_extent) + extent;
    	
    		//Creation of the type mpi_data_flow
    		ierr = MPI_Type_struct(MPI_STRUCT_ELTS + 1, blocklengths, displacements, types, &mpi_data_flow);
    		MPI_Type_commit(&mpi_data_flow);
    	
    		//Preparation of a 6th mpi new type that contains the two nodes that form a "remote" connection ==> struct Connection
    
    		types[0] = MPI_INT;
    		types[1] = MPI_INT;
    
    		blocklengths[0] = 1;
    		blocklengths[1] = 1;  
    		
    		displacements[0] = 0;
    		displacements[1] = extent;
    	
    		//Creation of the type mpi_geo
    		ierr = MPI_Type_struct(MPI_STRUCT_ELTS - 1, blocklengths, displacements, types, &mpi_connection);
    		MPI_Type_commit(&mpi_connection);
    	
    		//Preparation of a 7th mpi new type that contains partition details for a master ==> struct Partition	
    		for (int t = 0; t < 7; t++) {
    			types[t] = MPI_INT;
    			displacements[t] = t * extent;
    		}
    	
    		types[6] = MPI_FLOAT;
    		types[7] = MPI_FLOAT;
    		types[8] = MPI_FLOAT;
    		types[9] = MPI_INT;
    		types[10] = mpi_connection;
    	
    		displacements[7] = 6 * extent + float_extent;
    		displacements[8] = 6 * extent + 2 * float_extent;
    		displacements[9] = 6 * extent + 3 * float_extent;
    		displacements[10] = (6 + MAX_PARTITIONS) * extent + 3 * float_extent;
    	
    		for (int t = 0; t < 9; t++)
    			blocklengths[t]= 1;
    		
    	
    		blocklengths[9] = MAX_PARTITIONS;
    		blocklengths[10] = 3;
    	
    
    		//Creation of the type mpi_geo
    		ierr = MPI_Type_struct(MPI_STRUCT_ELTS + 8, blocklengths, displacements, types, &mpi_partition);
    		MPI_Type_commit(&mpi_partition);
    
    		mpi_types.mpi_master_buff = mpi_master_buff;
		mpi_types.mpi_data_flow = mpi_data_flow;
    
    
    		/************************/
    	
		if (commRank == 0) { // The coordinator code

#ifdef __timing__	

			//Database: creation and connection
			conn = mysql_init (NULL);
			
			if (mysql_real_connect (conn,host_name,user_name,password,db_name,port_num,socket_name,flags) == NULL){
  				fprintf(stderr, "Connection to database failed, error : %s\n", mysql_error(conn));
  			} 

			//Create table specific to this simulation (specific network size)
			sprintf(query,"DROP TABLE IF EXISTS %d_128_1_1_8", total_node_number);	
			process_query (conn,query);
	
			sprintf(query,"CREATE TABLE IF NOT EXISTS %d_128_1_1_8 (round SMALLINT UNSIGNED NOT NULL, master TINYINT UNSIGNED NOT NULL, round_len DECIMAL(10,6),MOB DECIMAL(10,6), CON DECIMAL(10,6), APP_OUT DECIMAL(10,6), APP_IN DECIMAL(10,6), PROTO_OUT DECIMAL(10,6), PROTO_IN DECIMAL(10,6), SND_M DECIMAL(10,6), SND_K DECIMAL(10,6), RCV_M DECIMAL(10,6), RCV_K DECIMAL(10,6), SYNC DECIMAL(10,6), FRWD_PKT MEDIUMINT UNSIGNED, FRWD_PKT_OUT MEDIUMINT UNSIGNED, PRIMARY KEY (round, master))", total_node_number);
			process_query (conn,query);

			mysql_close (conn);

#endif 

			int tag = 1 - commSize;		
			
			data_flow.nb_node = total_node_number;
			
			//Sending a part of work to each of the masters
				for (int proc = 1; proc < commSize; proc++) {
					struct Partition partition = simulation_parameters.distributed_simulation_config.partitions[proc - 1];
					
					// Send partitions to masters
					ierr = MPI_Send ( &partition, 1, mpi_partition, proc, 0, MPI_COMM_WORLD ) ;
				}
			
			if (write_descriptor != -3) {
		
				while (tag < 0) {
					for (int proc = 1; proc < commSize; proc++) {
					
						ierr = MPI_Recv( &data, 1, mpi_data_flow, MPI_ANY_SOURCE, MPI_ANY_TAG, MPI_COMM_WORLD, &status ) ;
						tag += status.MPI_TAG;
						int ofst; 
						if (status.MPI_TAG == 0) {
							ofst = simulation_parameters.distributed_simulation_config.partitions[status.MPI_SOURCE-1].offset;
							for (int nd = 0; nd < data.nb_node; nd++) {
								data_flow.geo[ofst + nd] = data.geo[nd];
							
								data_flow.geo[ofst + nd].p.x += 200 * (status.MPI_SOURCE - 1); 							
								data_flow.new_message[ofst + nd] = data.new_message[nd];	
							}
								
							data_flow.tour = data.tour;
						}
								
					}
					
					if (status.MPI_TAG == 0)
						if ( write( write_descriptor, &data_flow, sizeof(struct Data_Flow_Unit) ) < sizeof(struct Data_Flow_Unit) )
        	    			perror( "write random" );
            		
				}
			} else {
			
				for (int proc = 1; proc < commSize; proc++) {
					ierr = MPI_Recv( &data, 1, mpi_data_flow, MPI_ANY_SOURCE, MPI_ANY_TAG, MPI_COMM_WORLD, &status ) ;
				}
			}

		} else {
		
			for (int exec_nb = 0; exec_nb < number_of_executions; exec_nb++) {
				
				// Receiving partitions
				ierr = MPI_Recv( &partition, 1, mpi_partition, 0, MPI_ANY_TAG, MPI_COMM_WORLD, &status ) ;
				
				hipEventRecord(start, 0);

				switch (initial_distribution) {
			
					case RANDOM_DISTRIBUTION:
					
					partition.additional_time = 100 * partition.masters_number;
						
					performance = Random_sched(partition, 1, drop_probability, 
							(hiprandGenerator_t *) &gen, output_format, comm_masters, mpi_types);

	
					break;

					case GRID:
				
					performance = Static_grid(partition, 1, drop_probability,
							(hiprandGenerator_t *) &gen, output_format, comm_masters, mpi_types);
	
					break;
				}

				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsed, start, stop);

				total_elapsed += elapsed;

				if (elapsed > elapsed_max) {
					elapsed_max = elapsed;
				}
	
				if (elapsed < elapsed_min) {
					elapsed_min = elapsed;
				}
	
				total_dest += (float) performance.total_dest;
	
				total_forwarded += (float) performance.total_forwarded;

			}
		
			total_elapsed = total_elapsed / (float) number_of_executions; final_result [0] = total_elapsed ;	
			total_dest = total_dest / (float) number_of_executions;	final_result [1] = total_dest ;
			total_forwarded = total_forwarded / (float) number_of_executions; final_result [2] = total_forwarded;
			final_result [3] = elapsed_min; final_result [4] = elapsed_max;
			
			final_result [5] = performance.host_used_memory;
			final_result [6] = performance.device_used_memory;
			
			ierr = MPI_Send( final_result, 7, MPI_FLOAT, 0, 2, MPI_COMM_WORLD ) ;
			
		}
		
		if (commRank == 0){
	
			total_elapsed = 0;
			performance.host_used_memory = 0;
			performance.device_used_memory = 0;
		
			for (int iProc = 0; iProc < commSize - 1; iProc ++) {
				ierr = MPI_Recv( final_result, 7, MPI_FLOAT, MPI_ANY_SOURCE, 2, MPI_COMM_WORLD, &status ) ;
				
				if (final_result[4] > elapsed_max) {
					elapsed_max = final_result[4];
				}
	
				if (final_result[3] < elapsed_min) {
					elapsed_min = final_result[3];
				}
				
				total_dest += (float) final_result [1];
				total_forwarded += (float) final_result [2];
				total_elapsed += final_result [0];
				performance.host_used_memory = final_result[5];
				performance.device_used_memory = final_result[6];

				//printf("[C] Prc %d: Avr_elpsd = %f, loss = %f, tot_frwd = %d\n", status.MPI_SOURCE, final_result[0], 1 - (final_result[1]/(float)nb_tours), (int)final_result[2]);
			}
			total_elapsed /= (float) (commSize - 1);
		
	    	if (output_format)			
		    	final_data.node_number = total_node_number;
	    	else
		    	printf("%d ", total_node_number);

	 	   	if (output_format)
				final_data.loss = 1.0 - (total_dest / (float) nb_tours);
		   	else
				printf("%f ", 1.0 - (total_dest / (float) nb_tours));
				
			if (output_format)		
			    final_data.forwarded = (int)total_forwarded;
	    	else
		    	printf("%d ", (int)total_forwarded);

			if (output_format){		
				final_data.device_memory = performance.device_used_memory;
			    final_data.host_memory = performance.host_used_memory;
   			} else
		 	    printf("%f %f ", performance.device_used_memory, performance.host_used_memory);
	
			if (output_format)	
				final_data.average_time = total_elapsed;
		    else
			    printf("%f ", total_elapsed);
			
			if (output_format)		
			    final_data.min_time = elapsed_min;
			else
			    printf("%f ", elapsed_min);

			if (output_format)		
			    final_data.max_time = elapsed_max;
			else
			    printf("%f\n", elapsed_max);
			   
			
			if (write_descriptor != -3) {
			//printf("\n*** Final results ***\n\n* Node number: %d\n* Loss rate: %f\n* All forwarded packets: %d\n* Used memory [device, host]: [%d, %d]\n* Elapsed time (average): %f\n* Elapsed time (min): %f\n* Elapsed time (max): %f\n",final_data.node_number, final_data.loss, final_data.forwarded,final_data.device_memory, final_data.host_memory,final_data.average_time, final_data.min_time, final_data.max_time);
				int truc; 
				truc = write( write_descriptor, &final_data, sizeof(struct Final_Data) );
				if( truc < sizeof(struct Final_Data) )
        				perror( "write main" );

				if (::close (write_descriptor) == -1 ) /* we close the write desc. */
	    				perror( "close on write" );
			}
		}
	}
	
	ierr = MPI_Finalize() ;

	return 0;
}

void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(-1);
	}
}

