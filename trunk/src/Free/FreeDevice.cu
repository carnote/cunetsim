/*
 * FreeDevice.cu
 *
 *  Created on: Dec 7, 2011
 *      Author: benromdh
 */


#ifndef STRUCTURES_H_
#define STRUCTURES_H_
#include "../structures.h"
#endif /* STRUCTURES_H_ */


__host__ void Free_Device(struct Geo *Device_Geo, struct Geo2 *Device_Geo2, struct Cell *Device_Cell, struct Node *Device_Node, struct Buffer *Device_InPhy, struct Buffer *Device_OutPhy, struct RouterBuffer *Device_RouterBuffer, float *Device_Randx, float  *Device_Randy, float *Device_Randz)
{
	hipFree(Device_Geo);
	hipFree(Device_Geo2);
	hipFree(Device_Cell);
	hipFree(Device_Node);
	hipFree(Device_InPhy);
	hipFree(Device_OutPhy);
	hipFree(Device_RouterBuffer);
	hipFree(Device_Randx);
	hipFree(Device_Randy);
	hipFree(Device_Randz);
}
