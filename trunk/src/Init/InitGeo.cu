#include "hip/hip_runtime.h"
/*******************************************************************************

  Eurecom Cunetsim2
  Copyright(c) 2011 - 2012 Eurecom

  This program is free software; you can redistribute it and/or modify it
  under the terms and conditions of the GNU General Public License,
  version 2, as published by the Free Software Foundation.

  This program is distributed in the hope it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for
  more details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc.,
  51 Franklin St - Fifth Floor, Boston, MA 02110-1301 USA.

  The full GNU General Public License is included in this distribution in
  the file called "COPYING".

  Contact Information
  Cunetsim Admin: cunetsim@eurecom.fr
  Cunetsim Tech : cunetsim_tech@eurecom.fr
  Forums       : TODO
  Address      : Eurecom, 2229, route des crêtes, 06560 Valbonne Sophia Antipolis, France

*******************************************************************************/

/**
 * \file InitGeo.cu
 * \brief Functions necessary to initialize the space data
 * \author Bilel BR
 * \version 0.0.2
 * \date Nov 10, 2011
 */

#include <time.h>
#include <unistd.h>
#include <math.h>
#ifndef STRUCTURES_H_
#include "../structures.h"
#define STRUCTURES_H_
#endif /* STRUCTURES_H_ */
#ifndef INTERFACES_H_
#define INTERFACES_H_
#include "../interfaces.h"
#endif /* INTERFACES_H_ */
#include "../vars.h"



//#define _3D_


 /**
 * \fn __host__ void Init_geo(struct Cell *cell, struct Geo *geo, float *posrandx,
		float *posrandy, float *posrandz, float *vrandx, float *vrandy,
		float *vrandz, int node_number)
 * \brief initializes geographical properties of the nodes
 *
 * \param cell pointer to the cell distribution data
 * \param geo pointer to the space data
 * \param posrandx pointer to the table providing a random value for the x component of each node's position
 * \param posrandy pointer to the table providing a random value for the y component of each node's position
 * \param posrandz pointer to the table providing a random value for the z component of each node's position
 * \param vrandx pointer to the table providing a random value for the x component of each node's speed
 * \param vrandy pointer to the table providing a random value for the y component of each node's speed
 * \param vrandz pointer to the table providing a random value for the z component of each node's speed
 * \param node_number is the number of nodes in the simulation
 * \return void
 */
__host__ void Init_geo(struct Cell *cell, struct Geo *geo, float *posrandx,
		float *posrandy, float *posrandz, float *vrandx, float *vrandy,
		float *vrandz, int node_number) {

	int step_x = simulation_parameters.topology_config.area.geo_cell.step_x;
	int step_y = simulation_parameters.topology_config.area.geo_cell.step_y;
	int step_z = simulation_parameters.topology_config.area.geo_cell.step_z;

	int cell_size = simulation_parameters.topology_config.area.geo_cell.cell_size_m;
	int v_min = simulation_parameters.topology_config.mobility_parameters.moving_dynamics.min_speed_mps;
	int v_max = simulation_parameters.topology_config.mobility_parameters.moving_dynamics.max_speed_mps;
	int _3D = simulation_parameters.simulation_config._3D_is_activated;
	
	float energy = simulation_parameters.environment_config.init_energy;

	for (int i = 0; i < node_number; i++) {
		// We initialize the energy level for the node
		geo[i].energy = energy;
		
		// We initialize a node with a random position (but in the space because each coordinate is bounded by Step*Visibility)
		geo[i].p.x = (int) ((1 - posrandx[i]) * step_x * cell_size);
		geo[i].p.y = (int) ((1 - posrandy[i]) * step_y * cell_size);
		if ( _3D )
			geo[i].p.z = (int) ((1 - posrandz[i]) * step_z * cell_size);
		else
			geo[i].p.z = 0;

		// We initialize the speeds randomly but upper bounded by vmax and lower bounded by vmin
		geo[i].speedx = (int) ((v_max - v_min) * vrandx[i] + v_min);
		geo[i].speedy = (int) ((v_max - v_min) * vrandy[i] + v_min);///BBR14/11/2011
		if ( _3D )
			geo[i].speedz = (int) ((v_max - v_min) * vrandz[i] + v_min);
		else
			geo[i].speedz = 0;

		// The model of mobility is not the same for all the nodes, it is intialized randomly
		geo[i].cell_id = geo[i].p.x / cell_size + geo[i].p.y / cell_size
				* step_x + geo[i].p.z / cell_size * step_y * step_x;

		//printf("(%d,%d,%d,%d,%d)\n",i,geo[i].cell_id,geo[i].p.x,geo[i].p.y,geo[i].p.z);
		
		geo[i].CellPosition = 0; // ?
		geo[i].old_cell_id = geo[i].cell_id;
		cell[geo[i].cell_id].passage[i] = 1;
		cell[geo[i].cell_id].member[cell[geo[i].cell_id].size] = i;
		cell[geo[i].cell_id].size += 1;

		geo[i].neighbor_number = 0;
		//printf("Node: %d - GeoPos (x,y): (%d,%d) so I am in cell: %d\n",i,geo[i].p.x,geo[i].p.y,geo[i].cell_id);

	
	}
}

/*************************/
/*** Static grid model ***/
/*************************/


 /**
 * \fn __host__ void Init_geo_static_grid(struct Geo *geo, int node_number)
 * \brief initializes space data for the grid scenario
 *
 * \param geo pointer to the space data needed to set the initial position and the neighbors of each node according to the grid model
 * \param node_number is the number of nodes in the simulation
 * \return void
 */
__host__ void Init_geo_static_grid(struct Geo *geo, int node_number) {

	/*
	 *   The grid width and height are equal to the square root of node_number
	 */

	/* Verification of the value of node_number */
	if (pow(sqrt(node_number), 2) != node_number) {
		printf("Error: %d is not the square of an integer!\n",node_number);
		exit(0);
	}

	int grid_dimension = (int) sqrt(node_number);
	float energy = simulation_parameters.environment_config.init_energy;

	//printf("The grid dimension is: %d", grid_dimension);

	for (int i = 0; i < node_number; i++) {
		// We initialize the energy level for the node
		geo[i].energy = energy;

		// We initialize a node with a predefined position, but id does not have effect on the simulation
		geo[i].p.x = (i % grid_dimension) * 100;
		geo[i].p.y = (i / grid_dimension) * 100;
		geo[i].p.z = 0;

		// Looking for the neighbor_number
		geo[i].neighbor_number = 0;

		if (i - grid_dimension >= 0 && i - grid_dimension <= node_number - 1) {
			geo[i].neighbor_vector[geo[i].neighbor_number] = i - grid_dimension;
			geo[i].neighbor_number++;
		}

		if (i + grid_dimension >= 0 && i + grid_dimension <= node_number - 1) {
			geo[i].neighbor_vector[geo[i].neighbor_number] = i + grid_dimension;
			geo[i].neighbor_number++;
		}

		if (i - 1 >= 0 && i - 1 <= node_number - 1 && (i % grid_dimension != 0)) {
			geo[i].neighbor_vector[geo[i].neighbor_number] = i - 1;
			geo[i].neighbor_number++;
		}

		if (i + 1 >= 0 && i + 1 <= node_number - 1 && ((i + 1) % grid_dimension != 0)) {
			geo[i].neighbor_vector[geo[i].neighbor_number] = i + 1;
			geo[i].neighbor_number++;
		}

		/*printf("\nNode: %d - GeoPos (x,y,z): (%d,%d,%d) - neighbor_number: ", i,
		 geo[i].p.x, geo[i].p.y, geo[i].p.z);

		 for (int j = 0; j < geo[i].neighbor_number; j++)
		 printf("%d ", geo[i].Neighbor[j]);*/
	}
}
